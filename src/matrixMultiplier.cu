#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <Eigen/Dense>
#include <>
__global__ void cu_dot(Eigen::Vector3d *v1, Eigen::Vector3d *v2, double *out, size_t N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N)
    {
        out[idx] = v1[idx].dot(v2[idx]);
    }
    return;
}